
#include <hip/hip_runtime.h>
#include<cmath>
#include<cstdio>
#define M 2
#define N 2
#define K 2
#define n 4


 
__global__
void multiply(int*A,int*B,int*C)
{
int k,i,j,tmp;
for( i=0;i<M*K;i++)
{    printf("%d\n",A[i]);}

int I=blockIdx.x*blockDim.x+threadIdx.x;

int J=blockIdx.y*blockDim.y+threadIdx.y;
 
if( I < M  || j < N)

{

for( k=0;k<K;k++){
 
C[I*N+J]+= A[I*K+k]*B[k*N+J];

}

}
for (i = 0; i < M*N; i++)
 {   printf("[%d] =%d\n",i, C[i]);
}

}
int main(){
 
int A[M][K]={{1,2},{3,1}};
 int B[K][N]={{2,4},{5,2}};
 int C[M][N]={{0,0},{0,0}};
int* d_A;int* d_B;int* d_C;
 

hipMalloc(&d_A,n* sizeof(int));//let memory store that m*n space for you of size ints  
hipMalloc(&d_B,n* sizeof(int));
hipMalloc(&d_C,n* sizeof(int));

//copy Aand B FROM HOST TO DEVICE  

hipMemcpy(d_A, &A[0],n* sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_B, &B[0],n *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_C, &C[0],n*sizeof(int) , hipMemcpyHostToDevice);


multiply<<<1,1>>>(d_A,d_B,d_C);
//COPY RESULT BACK TO HOST
hipMemcpy(&C[0], d_C,n* sizeof(int), hipMemcpyDeviceToHost);


//printf("%d", C[0]);

hipFree(A);//TO FREE MEMORY

hipFree(B);
hipFree(C);
}
