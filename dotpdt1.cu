



#include <hip/hip_runtime.h>
#include<cmath>
#include<cstdio>
//#define BLOCKSIZE 1 


__global__
void dotproduct(int* A,int*B,int*C,int M,int N)
{
//printf("%d %d\n", A[0],A[1]);

//printf("%d %d\n", B[0],B[1]);

//printf("%d %d\n", C[0],C[1]);
M=2;

int I=blockIdx.x*blockDim.x+threadIdx.x;

int temp =0;

if( I < M  ){
for(int i=0;i<M;i++){
C[i]=A[i]*B[i];

temp+=C[i];


printf("matrix products:%d\n", C[i]);



}
printf("dot product is; %d\n",temp);

}

}
int 
main(){


int A[2]={1,2};
int B[2]={1,1};
int C[2]={0,0};
int* d_A;int* d_B;int* d_C;


int M=2;
int N=2;
//int K=2;


hipMalloc(&d_A, M *sizeof(int));//let memory store that m*n space for you of size ints 
hipMalloc(&d_B, M *sizeof(int));
hipMalloc(&d_C, sizeof(int));

//copy Aand B FROM HOST TO DEVICE 

hipMemcpy(d_A, &A[0],M *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_B, &B[0],M *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_C, &C[0],sizeof(int) , hipMemcpyHostToDevice);
dotproduct<<<1,1>>>(d_A,d_B,d_C,M,N);

//COPY RESULT BACK TO HOST
hipMemcpy(&C[0], d_C, sizeof(int), hipMemcpyDeviceToHost);



hipFree(A);//TO FREE MEMORY

hipFree(B);
hipFree(C);
}

                       
