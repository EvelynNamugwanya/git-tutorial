
#include <hip/hip_runtime.h>
#include<cmath> 
#include<cstdio>
//#define BLOCKSIZE 1 

__global__
void dotproduct(int* A,int*B,int*C,int M,int N,int K)
{
printf("%d %d\n", A[0],A[1]);

printf("%d %d\n", B[0],B[1]);

printf("%d %d\n", C[0],C[1]);


int I=blockIdx.x*blockDim.x+threadIdx.x;

int J=blockIdx.y*blockDim.y+threadIdx.y;
int temp =0;

if( I < M  || J < N){




//what is Bkj=B[k*N+J]; 
//int temp =0; 
for(int k=0;k<K;k++){
temp +=A[I*K+k]*B[k*N+J];
}
}
C[I*N+J]=temp;
//} 
printf("%d\n", C[I*N+J]);


//} 
}
int main(){

//int *A=(iny ) 
int A[2]={1,2};
int B[2]={1,1};
int C[2]={0,0};
int* d_A;int* d_B;int* d_C;
//int* A;int* B;int* C; 

int M=2;
int N=2;
int K=2;
//allocating space for variables on device 

hipMalloc(&d_A, M *sizeof(int));//let memory store that m*n space for you of size ints 
hipMalloc(&d_B, M *sizeof(int));
hipMalloc(&d_C, sizeof(int));
/*        //alocate space for variables on the host 
cudaMalloc(&A, M *N*sizeof(int));//let memory store that m*n space for you of size ints 
cudaMalloc(&B, M *N*sizeof(int)); 
cudaMalloc(&C, M *N*sizeof(int)); 
*/


//copy Aand B FROM HOST TO DEVICE 

hipMemcpy(d_A, &A[0],M *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_B, &B[0],M *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_C, &C[0],sizeof(int) , hipMemcpyHostToDevice);
dotproduct<<<1,1>>>(d_A,d_B,d_C,M,N,K );

//COPY RESULT BACK TO HOST
hipMemcpy(&C[0], d_C, sizeof(int), hipMemcpyDeviceToHost);
//printf("%d", C[0]);

hipFree(A);//TO FREE MEMORY

hipFree(B);
hipFree(C);
}



















