#include "hip/hip_runtime.h"

#include<cmath>
#include<cstdio>
#define M 2
#define N 2
#define K 2
__global__
void matrix_multiply(int* A,int*B,int*C)
{

//printf("%d %d\n", A[0],A[1]);

//printf("%d %d\n", C[0],C[1]);
for(int i=0;i<M*K;i++){    printf("%d\n",A[i]);}
for(int i=0;i<K*N;i++){    printf("%d\n",B[i]);}

int I=blockIdx.x*blockDim.x+threadIdx.x;

int J=blockIdx.y*blockDim.y+threadIdx.y;

if( I < M  || J < N)
{

for( i=0;i<M;i++){
for( j=0;j<N;j++){
for( k=0;k<K;k++){
C[I*N+J]=C[I*N+J]+ A[I*K+k]*B[k*N+J];

}
}

}

}
/********************************************************
row major 

 

what is Cij =C[I*n+J]; 


What is Aik=A[i*K+k]; 


what is Bkj=B[k*N+J]; 
*/**************************************************************************

}
for (i = 0; i < M * N; i++){
     printf("[%d] =%d\n",i, C[i]);

}
 
}
int main(){

int A[M*K]={1,2,3,1};
  int B[K*N]={2,4,5,2};
  int C[M*N]={0,0,0,0};
int* d_A;int* d_B;int* d_C;

//allocating space for variables on device 

hipMalloc(&d_A,M * sizeof(int));//let memory store that m*n space for you of size ints 
hipMalloc(&d_B, K * sizeof(int));
hipMalloc(&d_C, sizeof(int));


//copy A and B FROM HOST TO DEVICE 

hipMemcpy(d_A, &A[0],M* sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_B, &B[0],K *sizeof(int) , hipMemcpyHostToDevice);
hipMemcpy(d_C, &C[0],sizeof(int) , hipMemcpyHostToDevice);

matrix_multiply<<<1,1>>>(d_A,d_B,d_C);

//COPY RESULT BACK TO HOST
hipMemcpy(&C[0], d_C, sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(&C[1], d_C, sizeof(int), hipMemcpyDeviceToHost);
//printf("%d", C[0]);

hipFree(A);//TO FREE MEMORY

hipFree(B);
hipFree(C);
}

